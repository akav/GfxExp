#include "hip/hip_runtime.h"
﻿#define PURE_CUDA
#include "../restir_shared.h"

using namespace shared;

CUDA_DEVICE_KERNEL void performLightPreSampling() {
    uint32_t linearThreadIndex = blockDim.x * blockIdx.x + threadIdx.x;
    //uint32_t subsetIndex = linearThreadIndex / lightSubsetSize;
    uint32_t indexInSubset = linearThreadIndex % lightSubsetSize;
    PCG32RNG rng = plp.s->lightPreSamplingRngs[linearThreadIndex];

    // JP: 環境光テクスチャーが設定されている場合は一定の確率でサンプルする。
    //     ダイバージェンスを抑えるために、サブセットの最初とそれ以外で環境光かそれ以外のサンプリングを分ける。
    // EN: Sample an environmental light texture with a fixed probability if it is set.
    //     Separate sampling from the environmental light and the others to
    //     the beginning of the subset and the rest to avoid divergence.
    float probToSampleCurLightType = 1.0f;
    bool sampleEnvLight = false;
    if (plp.s->envLightTexture && plp.f->enableEnvLight) {
        if (plp.s->lightInstDist.integral() > 0.0f) {
            sampleEnvLight = indexInSubset < probToSampleEnvLight * lightSubsetSize;
            //sampleEnvLight = subsetIndex < probToSampleEnvLight * numLightSubsets;
            probToSampleCurLightType = sampleEnvLight ?
                probToSampleEnvLight : (1 - probToSampleEnvLight);
        }
        else {
            sampleEnvLight = true;
        }
    }

    PreSampledLight preSampledLight;
    sampleLight<false>(
        Point3D(0.0f),
        rng.getFloat0cTo1o(), sampleEnvLight, rng.getFloat0cTo1o(), rng.getFloat0cTo1o(),
        &preSampledLight.sample, &preSampledLight.areaPDensity);
    preSampledLight.areaPDensity *= probToSampleCurLightType;

    plp.s->lightPreSamplingRngs[linearThreadIndex] = rng;
    plp.s->preSampledLights[linearThreadIndex] = preSampledLight;
}



CUDA_DEVICE_KERNEL void performPerPixelRIS() {
    int2 launchIndex = make_int2(blockDim.x * blockIdx.x + threadIdx.x,
                                 blockDim.y * blockIdx.y + threadIdx.y);

    uint32_t curBufIdx = plp.f->bufferIndex;
    GBuffer0 gBuffer0 = plp.s->GBuffer0[curBufIdx].read(launchIndex);
    GBuffer1 gBuffer1 = plp.s->GBuffer1[curBufIdx].read(launchIndex);
    GBuffer2 gBuffer2 = plp.s->GBuffer2[curBufIdx].read(launchIndex);

    Point3D positionInWorld = gBuffer0.positionInWorld;
    Normal3D shadingNormalInWorld = gBuffer1.normalInWorld;
    Point2D texCoord(gBuffer0.texCoord_x, gBuffer1.texCoord_y);
    uint32_t materialSlot = gBuffer2.materialSlot;

    // JP: タイルごとに共通のライトサブセットを選択することでメモリアクセスのコヒーレンシーを改善する。
    // EN: Select a common light subset for each tile to improve memory access coherency.
    PCG32RNG rng = plp.s->rngBuffer.read(launchIndex);
    CUDA_SHARED_MEM uint32_t sm_perTileLightSubsetIndex;
    if (threadIdx.x == 0 && threadIdx.y == 0)
        sm_perTileLightSubsetIndex = mapPrimarySampleToDiscrete(rng.getFloat0cTo1o(), numLightSubsets);
    __syncthreads();
    uint32_t perTileLightSubsetIndex = sm_perTileLightSubsetIndex;
    const PreSampledLight* lightSubSet = &plp.s->preSampledLights[perTileLightSubsetIndex * lightSubsetSize];

    if (materialSlot == 0xFFFFFFFF)
        return;

    const MaterialData &mat = plp.s->materialDataBuffer[materialSlot];

    // TODO?: Use true geometric normal.
    Normal3D geometricNormalInWorld = shadingNormalInWorld;
    Vector3D vOut = plp.f->camera.position - positionInWorld;
    float frontHit = dot(vOut, geometricNormalInWorld) >= 0.0f ? 1.0f : -1.0f;

    BSDF bsdf;
    bsdf.setup(mat, texCoord, 0.0f);
    ReferenceFrame shadingFrame(shadingNormalInWorld);
    positionInWorld = offsetRayOriginNaive(positionInWorld, frontHit * geometricNormalInWorld);
    float dist = length(vOut);
    vOut /= dist;
    Vector3D vOutLocal = shadingFrame.toLocal(vOut);

    uint32_t curResIndex = plp.currentReservoirIndex;
    Reservoir<LightSample> reservoir;
    reservoir.initialize();

    // JP: Unshadowed ContributionをターゲットPDFとしてStreaming RISを実行。
    // EN: Perform streaming RIS with unshadowed contribution as the target PDF.
    float selectedTargetDensity = 0.0f;
    uint32_t numCandidates = 1 << plp.f->log2NumCandidateSamples;
    for (int i = 0; i < numCandidates; ++i) {
        uint32_t lightIndex = mapPrimarySampleToDiscrete(rng.getFloat0cTo1o(), lightSubsetSize);
        const PreSampledLight &preSampledLight = lightSubSet[lightIndex];

        // JP: 候補サンプルを生成して、ターゲットPDFを計算する。
        //     ターゲットPDFは正規化されていなくても良い。
        // EN: Generate a candidate sample then calculate the target PDF for it.
        //     Target PDF doesn't require to be normalized.
        RGB cont = performDirectLighting<ReSTIRRayType, false>(
            positionInWorld, vOutLocal, shadingFrame, bsdf,
            preSampledLight.sample);
        float targetDensity = convertToWeight(cont);

        // JP: 候補サンプル生成用のPDFとターゲットPDFは異なるためサンプルにはウェイトがかかる。
        // EN: The sample has a weight since the PDF to generate the candidate sample and the target PDF are
        //     different.
        float weight = targetDensity / preSampledLight.areaPDensity;
        if (reservoir.update(preSampledLight.sample, weight, rng.getFloat0cTo1o()))
            selectedTargetDensity = targetDensity;
    }

    // JP: 現在のサンプルが生き残る確率密度の逆数の推定値を計算する。
    // EN: Calculate the estimate of the reciprocal of the probability density that the current sample survives.
    float recPDFEstimate = reservoir.getSumWeights() / (selectedTargetDensity * reservoir.getStreamLength());
    if (!isfinite(recPDFEstimate)) {
        recPDFEstimate = 0.0f;
        selectedTargetDensity = 0.0f;
    }

    ReservoirInfo reservoirInfo;
    reservoirInfo.recPDFEstimate = recPDFEstimate;
    reservoirInfo.targetDensity = selectedTargetDensity;

    plp.s->rngBuffer.write(launchIndex, rng);
    plp.s->reservoirBuffer[curResIndex][launchIndex] = reservoir;
    plp.s->reservoirInfoBuffer[curResIndex].write(launchIndex, reservoirInfo);
}
